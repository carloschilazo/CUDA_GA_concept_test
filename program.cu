#include <iostream>
#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime.h>
using namespace std;

//Kernel to get exponent, only positives
__device__ void calculate_exponent(int base,int exponent,long &result){
	result = 1;
	if(exponent==0){
		return;
	}
	for(int counter=1;counter<=exponent;counter++){
		result *= base;
	}
}

// Kernel to fill the array with somethin', in this case its just the position but works
__global__ void fill_array(int *input, int totalSizeOfArray, int individualsPerThread, int number_genes, int *randomNumbers)
{
	int startingPosition = threadIdx.y * (number_genes*individualsPerThread);
	for(int counter=0;counter<(individualsPerThread*number_genes);counter++){
		if(counter+startingPosition>=totalSizeOfArray){
			return;
		}

		input[counter+startingPosition] = randomNumbers[counter+startingPosition];
	}

}
// Kernel to evaluate an individual
__global__ void evaluate(int *input, int totalSizeOfArray, int number_genes, int individualsPerThread, long *scores){

	int startingPosition = threadIdx.y * (number_genes*individualsPerThread);
	int startingPosition_scores = threadIdx.y * individualsPerThread;
	long acumulated = 0;
	long temp = 0;
	for(int counter_individuals=0;counter_individuals<individualsPerThread;counter_individuals++){
		if(startingPosition + (counter_individuals*number_genes) >= totalSizeOfArray){
			return;
		}
		for(int counter_gene=0;counter_gene<number_genes;counter_gene++){
			int base = startingPosition + (counter_individuals*number_genes) + counter_gene;
			calculate_exponent(input[base],(number_genes-1)-counter_gene,temp);
			acumulated += temp;
		}
		scores[(threadIdx.y*individualsPerThread)+counter_individuals] = acumulated;
		
		acumulated=0;
	}

}





// main routine that executes on the host
int main(void)
{

	const int number_genes = 10;
	const int number_individuals = 1000000;

	int *population_array_host = new int[number_genes*number_individuals];
	int *population_array_device;

	long *score_array_host = new long[number_individuals];
	long *score_array_device;

	int *random_numbers_host = new int[number_genes*number_individuals];
	int *random_numbers_device;
	
	//we need to initialize the population array
	//must be done randomly
	//we calculate the number of threads required to fill the array in parallel
	int individuals_per_thread = 2000;
	int number_of_threads = number_individuals/individuals_per_thread + (number_individuals%individuals_per_thread == 0 ? 0:1);
	//we now randomly fill the random numbers array
	srand ( time(NULL));
	for(int contador=0;contador<number_genes*number_individuals;contador++){
		random_numbers_host[contador] = ( rand()  % 10 );
	}
	//we move the random numbers array to device
	size_t memory_for_random_numbers = number_genes*number_individuals*sizeof(int);
	hipMalloc((void **) &random_numbers_device, memory_for_random_numbers);
	hipMemcpy(random_numbers_device, random_numbers_host, memory_for_random_numbers, hipMemcpyHostToDevice);

	//we zero-ise the scores
	for(int contador=0;contador<number_individuals;contador++){
		score_array_host[contador] = 0;
	}
	//we move the scores array to device
	size_t memory_for_scores = number_individuals*sizeof(long);
	hipMalloc((void **) &score_array_device, memory_for_scores);
	hipMemcpy(score_array_device, score_array_host, memory_for_scores, hipMemcpyHostToDevice);

	//now we must launch 1 block with dimensions: x=1,y=number_of_threads, we define them
	dim3 grid_fill(1,1);
	dim3 block_fill(1,number_of_threads);
	//we now allocate memory in device
	size_t memory_for_population = number_genes*number_individuals*sizeof(int);
	hipMalloc((void **) &population_array_device, memory_for_population);
	//we now launch the kernel for populating
	fill_array <<< grid_fill, block_fill >>> (population_array_device, number_genes * number_individuals, individuals_per_thread, number_genes,random_numbers_device);

	//we now launch the kernel for evaluating
	evaluate <<< grid_fill, block_fill >>> (population_array_device, number_genes * number_individuals, number_genes, individuals_per_thread,score_array_device);

	hipMemcpy(population_array_host, population_array_device, memory_for_population, hipMemcpyDeviceToHost);
	hipMemcpy(score_array_host, score_array_device, memory_for_scores, hipMemcpyDeviceToHost);

	

	
	///END, move back to host, print PopulationArray

	for(int contador=0;contador<number_genes*number_individuals;contador++){
		if(contador%number_genes==0 && contador > 0){
			cout << endl;
		}
		cout << population_array_host[contador] << "-";

	}
	cout << endl;
	cout << "----";
	cout << endl;
	
	for(int contador=0;contador<number_individuals;contador++){
		cout << score_array_host[contador] << endl;
	}
	




}
